#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<stdio.h>
#include"md5.cu"
#include"gpu.h"
#define NUM_BLOCKS 128
#define THREADS_PER_BLOCK 256
#define HASH_PER_THREAD 512
#define MAX_N 12
#define MAX_PREF 10
#define MAX_SUFF 10
#define MAX_LEN 32
#define uint unsigned int

__device__ __constant__ char prefix[MAX_PREF];
__device__ __constant__ int lenPref;
__device__ __constant__ char suffix[MAX_SUFF];
__device__ __constant__ int lenSuf;
__device__ __constant__ char start[MAX_N];

__device__ void dincrement(unsigned char *word, int len, int charsetlen, int inc){
	int i=len-1;
	while(inc > 0 && i >= 0){
		int add = inc + word[i];
		word[i] = add%charsetlen;
		inc = add/charsetlen;
		i--;
	}
}

__global__ void kernel(int len, int numThreads, int md5PerThread, char *res){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	/*for(int i=0; i<md5PerThread; i++){
		if(tid == 0 && i%1 ==0)
		printf("%d %d\n", i, md5PerThread);
	}*/
	printf("GPU: %d %s %s %d %d %s\n", tid, prefix, suffix, lenPref, lenSuf, start);
	return;
}

__global__ void calcKernel(uint h0, uint h1, uint h2, uint h3, int len,
														 int numThreads, int md5PerThread, char *res,
 														 char *charset, int charsetlen){
	int tid = threadIdx.x + blockDim.x * blockIdx.x, lp = lenPref, ls = lenSuf;
	unsigned char myBrut[MAX_LEN], word[MAX_LEN];
	for(int i=0; i<lp; i++){
		word[i] = prefix[i];
	}
	for(int i=0; i<ls; i++){
		word[i + lp + len] = suffix[i];
	}
	for(int i=0; i<len; i++){
		myBrut[i] = start[i];
	}
	dincrement(myBrut, len, charsetlen, tid);
	uint mh0, mh1, mh2, mh3;
	int i;
	for(i=0; i<md5PerThread; i++){
		for(int j=0; j<len; j++){
			word[j+lp] = charset[myBrut[j]];
		}
		mh0 = 0, mh1 = 0, mh2 = 0, mh3 = 0;
		md5_vfy(word, lp+ls+len, &mh0, &mh1, &mh2, &mh3);
		if(h0 == mh0 && h1 == mh1 && h2 == mh2 && h3 == mh3){
			printf("%s\n", word);
			for(int j=0; j<len; j++){
				res[j] = charset[myBrut[j]];
			}
		}
		dincrement(myBrut, len, charsetlen, numThreads);
	}
	return;
}

int increment(unsigned char *word, int len, int charsetlen, int inc){
	int i=len-1;
	while(inc > 0 && i >= 0){
		int add = inc + word[i];
		word[i] = add%charsetlen;
		inc = add/charsetlen;
		i--;
	}
	return inc > 0;
}

int HC_calc(char *hash, char *pref, int lp, char *suff, int ls, char *charset, int lc, int len){
	int fin = 0;
	char *d_res, *d_charset, *st = (char*)malloc(len + 1);
	hipMemcpyToSymbol(HIP_SYMBOL(prefix), pref, lp, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(suffix), suff, ls, 0, hipMemcpyHostToDevice);
	
	hipMemcpyToSymbol(HIP_SYMBOL(lenPref), &lp, 4, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(lenSuf), &ls, 4, 0, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_res, len);
	hipMalloc((void**)&d_charset, lc);
	HC_res = (char*)malloc(len);
	for(int i=0; i<len; i++)
	HC_res[i] = 0;
	for(int i=0; i<len; i++)
	st[i] = 0;
	st[len] = '\x00';
	hipMemcpy(d_res, HC_res, len, hipMemcpyHostToDevice);
	hipMemcpy(d_charset, charset, lc, hipMemcpyHostToDevice);
	uint iH0, iH1, iH2, iH3;
	md5_to_ints((unsigned char*)hash, &iH0, &iH1, &iH2, &iH3);
	while(fin == 0){
		hipMemcpyToSymbol(HIP_SYMBOL(start), st, len, 0, hipMemcpyHostToDevice);
		calcKernel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(iH0, iH1, iH2, iH3, len, NUM_BLOCKS*THREADS_PER_BLOCK, HASH_PER_THREAD, d_res, d_charset, lc);
		fin = increment((unsigned char*)st, len, lc, NUM_BLOCKS*THREADS_PER_BLOCK*HASH_PER_THREAD);
		//kernel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(len, NUM_BLOCKS*THREADS_PER_BLOCK, HASH_PER_THREAD, d_res);
		hipMemcpy(HC_res, d_res, len, hipMemcpyDeviceToHost);
		printf("CALCULATED: ");
		for(int i=0; i<len; i++)
		printf("%d ", st[i]);
		printf("\n");
		for(int i=0; i<len; i++){
			if(HC_res[i] != 0){
				return 1;
			}
		}
	}
	hipFree(d_res);
	hipFree(d_charset);
	free(HC_res);
	return 0;
}

char *getRes(){
	return HC_res;
}
